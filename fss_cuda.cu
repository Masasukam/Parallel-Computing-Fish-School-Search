#include "hip/hip_runtime.h"
#include "fss.h"
#include <cmath>
#include <random>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define THRUST_IGNORE_DEPRECATED_CPP_DIALECT
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

#define NUM_THREADS 256

#define MIN(x, y) (x < y ? x : y)
#define SQR(x) ((x) * (x))

// when set to 1, the program will print some info for debugging
#define debug_mode 0

int t;
int blks; // number of blocks
double cfvalnx, cfvalny, cfvald; // numerator and denominator of collective movement

// benchmark function
__device__
double f(double x, double y, int t) {
    // return sin(sqrt(x * x + y * y));
    // return - ((x - 5) * (x - 5)) - ((y - 5) * (y - 5));
    return (-1 + 2.0 * t / nsteps) * (((x - 5) * (x - 5)) + ((y - 5) * (y - 5)));
    // return sin(0.01 * x * x + 0.005 * y * y - 0.05 * x + 2 * sin(0.01 * t));
    // return sin(0.01 * x * x + 0.005 * y * y - 0.05 * x + 2 * sin(0.005 * t));
}

// Function to calculate distance between two fishes
__device__
double distanceSquared(const fish_t &fish1, const fish_t &fish2) {
    return SQR(fish1.x - fish2.x) + SQR(fish1.y - fish2.y);
}

// Todo0: individual fish distances
__global__
void individual_move(fish_t* fish, int nfish, int t, double &cfvalnx, double &cfvalny, double &cfvald) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= nfish) return;

    hiprandState state; // random number generator
    hiprand_init(clock64(), tid, 0, &state);

    fish[tid].fval = f(fish[tid].x, fish[tid].y, t);
    fish[tid].ax = fish[tid].ay = 0;
    
    double next_fval = f(fish[tid].x, fish[tid].y, t);
    double newvx, newvy;
    for (int j = 0; j < retrynum; ++ j) {
        newvx = hiprand_uniform(&state) * 2.0f - 1.0f;
        newvy = hiprand_uniform(&state) * 2.0f - 1.0f;
        if (f(fish[tid].x + newvx * dt, fish[tid].y + newvy * dt, t) > next_fval) {
            printf(".\n");
            next_fval = f(fish[tid].x + fish[tid].vx * dt, fish[tid].y + fish[tid].vy * dt, t);
            fish[tid].vx = newvx;
            fish[tid].vy = newvy;
        }
    }

    cfvalnx += fish[tid].vx * (next_fval - fish[tid].fval);
    cfvalny += fish[tid].vy * (next_fval - fish[tid].fval);
    cfvald += fabs(next_fval - fish[tid].fval);

    // Todo 1: Add weight features
    // fish[tid].weight += (next_fval - fish[tid].fval) / fabs(next_fval - fish[tid].fval); // problematic
    // fish[tid].weight = fmax(1, fish[tid].weight);
    // fish[tid].weight = fmin(Wscale, fish[tid].weight);
}


// Todo2: within group distance
__global__
void collective_move(fish_t* fish, int nfish) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= nfish) return;

    double cutoffSquared = SQR(cutoff);
    double totalAdjustmentX = 0.0 - cutoff;
    double totalAdjustmentY = 0.0 - cutoff;
    int count = -1;

    for (int j = 0; j < nfish; ++ j) {
        double distSquared = distanceSquared(fish[tid], fish[j]);
        if (distSquared < cutoffSquared) {
            totalAdjustmentX += cutoff - fish[tid].x + fish[j].x;
            totalAdjustmentY += cutoff - fish[tid].y + fish[j].y;
            count ++ ;
        }
    }

    if (count > 0) {
        fish[tid].ax += totalAdjustmentX / count;
        fish[tid].ay += totalAdjustmentY / count;
    }
}

// Integrate the ODE
__global__
void move_fish(fish_t* fish, int nfish, double size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= nfish) return;

    fish[tid].vx += fish[tid].ax * dt;
    fish[tid].vx = MIN(fish[tid].vx, vmax);
    fish[tid].vy += fish[tid].ay * dt;
    fish[tid].vy = MIN(fish[tid].vy, vmax);

    fish[tid].x += fish[tid].vx * dt;
    fish[tid].y += fish[tid].vy * dt;


    // Bounce from walls
    while (fish[tid].x < 0 || fish[tid].x > size) {
        fish[tid].x = fish[tid].x < 0 ? -fish[tid].x : 2 * size - fish[tid].x;
        fish[tid].vx = -fish[tid].vx;
    }

    while (fish[tid].y < 0 || fish[tid].y > size) {
        fish[tid].y = fish[tid].y < 0 ? -fish[tid].y : 2 * size - fish[tid].y;
        fish[tid].vy = -fish[tid].vy;
    }
}


void init_simulation(fish_t* fish, int nfish, double size) {
	// You can use this space to initialize static, global data objects
    // that you may need. This function will be called once before the
    // algorithm begins. Do not do any fish simulation here
    t = 0;
    blks = (nfish + NUM_THREADS - 1) / NUM_THREADS;

    #if debug_mode & 1
    setvbuf( stdout, NULL, _IONBF, 0 );
    #endif
}

void simulate_one_step(fish_t* fish, int nfish, double size) {
    ++ t;
	cfvalnx = cfvalny = cfvald = 0;

    individual_move<<<blks, NUM_THREADS>>>(fish, nfish, t, cfvalnx, cfvalny, cfvald);

    hipDeviceSynchronize();

    #if debug_mode & 1
    printf("here 1\n");
    #endif

    collective_move<<<blks, NUM_THREADS>>>(fish, nfish);

    hipDeviceSynchronize();

    #if debug_mode & 1
    printf("here 2\n");
    #endif

    move_fish<<<blks, NUM_THREADS>>>(fish, nfish, size);

    #if debug_mode & 1
    printf("here 3\n");
    #endif
}